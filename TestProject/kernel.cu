#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

hipError_t mulMatrixOnVectorWithCuda(double *result, const double *mat, const double *vec, int matX, int matY);

__global__ void addKernel(int *c, const int *a, const int *b)//global - ����������� �� ����������
{
	//griddim - ����������� ����� � ������
	//blockdim -����������� ����� � ������
	//blockIdx
	//threadIdx
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

//__global__ void myAddKernel(int *c, const int *a, const int *b)//TODO:��������
//{
//
//}

__global__ void mulMatrixOnVectorKernel(double *result, const double *mat, const double *vec, int matX, int matY)
{
	int i = threadIdx.x, j = threadIdx.y;
	result[i] += mat[i*j] * vec[j];
}

void printArray(int size, int *a)
{
	printf("{");
	for (size_t i = 0; i < size - 1; i++)
	{
		printf("%d,", a[i]);
	}
	printf("%d}", a[size - 1]);
}

int main()
{
	const int arraySize = 100, blockSize = 16;
	int gridsize = arraySize / blockSize;
	/*
	int* a = new int[arraySize];
	int* b = new int[arraySize];
	int* c = new int[arraySize];
	for (size_t i = 0; i < arraySize; i++)
	{
		a[i] = (i + 1) * 3;
		b[i] = (i + 1) * 7;
		c[i] = 0;
	}

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printArray(arraySize, a);
	printf(" + ");
	printArray(arraySize, b);
	printf(" = ");
	printArray(arraySize, c);
	

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//	c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	*/
	const int matX = 25, matY = 25;
	double* matrix = new double[matX*matY], double* vec=new double[matY], double* result;

	clock_t t = clock();
	hipError_t cudaStatus = mulMatrixOnVectorWithCuda(result, matrix, vec,matX,matY);
	t = clock() - t;
	printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;//��������� �� ����������
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);//call from HOST on DEVICE

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

hipError_t mulMatrixOnVectorWithCuda(double * result, const double * mat, const double * vec, int matX, int matY)
{
	int *dev_mat = 0;//��������� �� ����������
	int *dev_vec = 0;
	int *dev_res = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_res, matX * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_mat, matX * matY * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_vec, matY * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_mat, mat, matX * matY * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_vec, vec, matY * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	mulMatrixOnVectorKernel <<<1, matX*matY >>> (*dev_res, *dev_mat, *dev_vec, matX, matY);//call from HOST on DEVICE

													 // Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {	
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_res, matX * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_mat);
	hipFree(dev_vec);
	hipFree(dev_res);

	return cudaStatus;
}

hipError_t myAddWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;//��������� �� ����������
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >> > (dev_c, dev_a, dev_b);//call from HOST on DEVICE

													 // Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
